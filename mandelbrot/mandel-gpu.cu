#include "hip/hip_runtime.h"
/* 
* File:   mandel.c
* Author: Antonio Lechuga
*
* Created on Día 9999 de la cuarentena COVID19
*/

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
//PP#include <hip/hip_runtime.h>

# define POINTS_PER_DIM 1024
# define MAX_ITER 2000

// Defining complex type
typedef struct complex_ {
    double real;
    double imag;
} complex, *Pcomplex;

// Getting new complex number
complex new_complex(double real, double imag) {
    Pcomplex complex_ptr = (Pcomplex)malloc(sizeof(complex));
    complex_ptr->real = real;
    complex_ptr->imag = imag;
    return *complex_ptr;
}

/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

// Mandelbrot generation kernel
__global__ void generate_mandelbrot(complex *in, int *out, complex z, int i_size, int max_iter) {
    
    // calculating indices
    int id_r = blockIdx.x * blockDim.x + threadIdx.x;
    int id_i = blockIdx.y * blockDim.y + threadIdx.y;

    // initial values
    complex c = in[id_i * i_size + id_r];
    int result = 1;
    double temp_real;
    double abs_value;

    // determining if c is part of mandelbrot set
    for (int i = 0; i < max_iter; i++) {
        // squaring z and adding c
        temp_real = z.real;
        z.real = (z.real * z.real) - (z.imag * z.imag) + c.real;
        z.imag = 2 * temp_real * z.imag + c.imag;
        // calculating abs value
        abs_value = sqrt((z.real * z.real) + (z.imag * z.imag));
        if (abs_value > 2.0) {
            result = 0;
            break;
        }
    }
    out[id_i * i_size + id_r] = result;
}

int main(int argc, char** argv) {

    // parsing input
    int r_points, i_points;
    if (argc < 2) {
        r_points = POINTS_PER_DIM;
        i_points = POINTS_PER_DIM;
    } else if (argc < 3) {
        r_points = 1 << atoi(argv[1]);
        i_points = 1 << atoi(argv[1]);
    } else {
        printf("Usage: mandel-gpu <log(xdim)> <log(ydim)>\n");
        exit(-1);
    }

    // initialization
    time_t t1, t2, t3, t4;
    double max = 2.0;
    double min = -2.0;
    int array_size = r_points * i_points;
    int num_outside = 0;
    double dR = (max - min) / r_points;
    double dI = (max - min) / i_points;
    complex z;
    z.real = 0.0;
    z.imag = 0.0;

    // calculating sizes
    size_t size_input = array_size * sizeof(complex);
    size_t size_output = array_size * sizeof(int);

    // pointers
    complex *h_input;                   // CPU
    complex *d_input;                   // CPU
    int *h_output;                      // GPU
    int *d_output;                      // GPU

    // allocating space in CPU
    h_input = (complex *) malloc(size_input);
    h_output = (int *) malloc(size_output);

    // allocating space in GPU
    hipMalloc((void **) &d_input, size_input);
    hipMalloc((void **) &d_output, size_output);


    // generating input
    printf("Generating input...\n");
    for (int i = 0; i < i_points; i++) {
        for (int j = 0; j < r_points; j++) {
            double real_part = min + dR * j;
            double imag_part = max - dI * i;
            h_input[i_points * i + j] = new_complex(real_part, imag_part);
        }
    }

    // copying from CPU to GPU
    hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);

    // executing kernels
    t1 = time(NULL);
    int n_threads = 32;
    int n_blocks_r = r_points / n_threads;
    int n_blocks_i = i_points / n_threads;
    dim3 dimBlock(n_threads, n_threads);
    dim3 dimGrid(n_blocks_r, n_blocks_i);

    generate_mandelbrot<<<dimGrid, dimBlock>>>(d_input, d_output, z, i_points, MAX_ITER);
    
    // waiting for threads
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    // timing execution
    t2 = time(NULL);

    // copying back to CPU
    hipMemcpy(h_output, d_output, size_output, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy");

    // adding all values outside mandelbrot set
    t3 = time(NULL);
    for (int i = 0; i < array_size; i++) {
        num_outside += h_output[i];
    }
    t4 = time(NULL);

    printf("Execution time: %f sec\n", difftime(t2, t1) + difftime(t4, t3));

    // number of points outside, area and error
    printf("The number of points outside is: %d\n", num_outside);
    float area = (2.0 * max) * (2.0 * max) * (double)(array_size - num_outside) / (double)(array_size);
    float error = area / (double)r_points;
    printf("Area of Mandlebrot set is: %12.8f +/- %12.8f\n", area, error);

    // generating pmg image
    printf("Generating image...\n");
    FILE *fp;
    fp = fopen("mandelbrot-fractal-gpu.pgm", "w");
    fputs("P2 \n", fp);
    fprintf(fp, "%d %d \n", i_points, r_points);
    fputs("1 \n", fp);
    for (int i = 0; i < i_points; i++) {
        for (int j = 0; j < r_points; j++) {
            fprintf(fp, "%d ", h_output[i * i_points + j]);
        }
        fputs("\n", fp);
    }
    fclose(fp);

    // freeing memory
    printf("Freeing memory...\n");
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    printf("Done!\n");
    return 0;
}

// Utility function to check for and report CUDA errors
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}