#include "hip/hip_runtime.h"
/* 
* File:   mandel.c
* Author: Antonio Lechuga
*
* Created on Día 9999 de la cuarentena COVID19
*/

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
//PP#include <hip/hip_runtime.h>

# define POINTS_PER_DIM 1024
# define MAX_ITER 2000

// Defining complex type
typedef struct complex_ {
    double real;
    double imag;
} complex, *Pcomplex;

// Getting new complex number
complex new_complex(double real, double imag) {
    Pcomplex complex_ptr = (Pcomplex)malloc(sizeof(complex));
    complex_ptr->real = real;
    complex_ptr->imag = imag;
    return *complex_ptr;
}

/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

// Mandelbrot generation kernel
__global__ void generate_mandelbrot(complex *in, int *out, complex z, int i_size, int max_iter) {
    
    // calculating indices
    int id_r = blockIdx.x * blockDim.x + threadIdx.x;
    int id_i = blockIdx.y * blockDim.y + threadIdx.y;

    // initial values
    complex c = in[id_i * i_size + id_r];
    int result = 1;
    double temp_real;
    double abs_value;

    // determining if c is part of mandelbrot set
    for (int i = 0; i < max_iter; i++) {
        // squaring z and adding c
        temp_real = z.real;
        z.real = (z.real * z.real) - (z.imag * z.imag) + c.real;
        z.imag = 2 * temp_real * z.imag + c.imag;
        // calculating abs value
        abs_value = sqrt((z.real * z.real) + (z.imag * z.imag));
        if (abs_value > 2.0) {
            result = 0;
            break;
        }
    }
    out[id_i * i_size + id_r] = result;

    __syncthreads();
    // calculating number of elements outside of mandelbrot set
    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
        int num_inside = 0;
        for (int i = 0; i < i_size * i_size; i++) {
            num_inside += out[i];
        }
        float area = 16.0 * (double)(num_inside) / (double)(i_size * i_size);
        float error = area / (double)i_size;
        printf("The number of points outside is: %d\n", i_size * i_size - num_inside);
        printf("Area of Mandlebrot set is: %12.8f +/- %12.8f\n", area, error);
    }
}

int main(int argc, char** argv) {

    // parsing input
    int r_points, i_points;
    if (argc < 2) {
        r_points = POINTS_PER_DIM;
        i_points = POINTS_PER_DIM;
    } else if (argc < 3) {
        r_points = 1 << atoi(argv[1]);
        i_points = 1 << atoi(argv[1]);
    } else {
        printf("Usage: mandel-gpu <log(xdim)> <log(ydim)>\n");
        exit(-1);
    }

    // initialization
    time_t t1, t2;
    double max = 2.0;
    double min = -2.0;
    int array_size = r_points * i_points;
    // int num_outside = 0;
    double dR = (max - min) / r_points;
    double dI = (max - min) / i_points;
    complex z;
    z.real = 0.0;
    z.imag = 0.0;

    // calculating sizes
    size_t size_input = array_size * sizeof(complex);
    size_t size_output = array_size * sizeof(int);

    // pointers
    complex *h_input;                   // CPU
    complex *d_input;                   // CPU
    int *h_output;                      // GPU
    int *d_output;                      // GPU

    // allocating space in CPU
    h_input = (complex *) malloc(size_input);
    h_output = (int *) malloc(size_output);

    // allocating space in GPU
    hipMalloc((void **) &d_input, size_input);
    hipMalloc((void **) &d_output, size_output);


    // generating input
    printf("Generating input...\n");
    for (int i = 0; i < i_points; i++) {
        for (int j = 0; j < r_points; j++) {
            double real_part = min + dR * j;
            double imag_part = max - dI * i;
            h_input[i_points * i + j] = new_complex(real_part, imag_part);
        }
    }

    // copying from CPU to GPU
    hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);

    // executing kernels
    t1 = time(NULL);
    int n_threads = 32;
    int n_blocks_r = r_points / n_threads;
    int n_blocks_i = i_points / n_threads;
    dim3 dimBlock(n_threads, n_threads);
    dim3 dimGrid(n_blocks_r, n_blocks_i);

    generate_mandelbrot<<<dimGrid, dimBlock>>>(d_input, d_output, z, i_points, MAX_ITER);
    
    // waiting for threads
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    // timing execution
    t2 = time(NULL);
    printf("Execution time: %f sec\n", difftime(t2, t1));

    // copying back to CPU
    hipMemcpy(h_output, d_output, size_output, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy");

    // generating pmg image
    printf("Generating image...\n");
    FILE *fp;
    fp = fopen("mandelbrot-fractal-gpu.pgm", "w");
    fputs("P2 \n", fp);
    fprintf(fp, "%d %d \n", i_points, r_points);
    fputs("1 \n", fp);
    for (int i = 0; i < i_points; i++) {
        for (int j = 0; j < r_points; j++) {
            fprintf(fp, "%d ", h_output[i * i_points + j]);
        }
        fputs("\n", fp);
    }
    fclose(fp);

    // freeing memory
    printf("Freeing memory...\n");
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    printf("Done!\n");
    return 0;
}

// Utility function to check for and report CUDA errors
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}