#include "hip/hip_runtime.h"
/*
 * 
 * Programa de Introducción a los conceptos de CUDA
 * 
 *
 * 
 * 
 */

#include <stdio.h>
#include <stdlib.h>

/* Declaración de métodos/


/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

/* Kernel para sumar dos vectores en un sólo bloque de hilos */
__global__ void vect_add(int *d_a, int *d_b, int *d_c)
{
    /* Part 2B: Implementación del kernel para realizar la suma de los vectores en el GPU */
    int idx = threadIdx.x;
    d_c[idx] = d_a[idx] + d_b[idx];
}

/* Versión de múltiples bloques de la suma de vectores */
__global__ void vect_add_multiblock(int *d_a)
{
    /* Part 2C: Implementación del kernel pero esta vez permitiendo múltiples bloques de hilos. */
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    d_c[idx] = d_a[idx] + d_b[idx];
}

/* Numero de elementos en el vector */
#define ARRAY_SIZE 256

/*
 * Número de bloques e hilos
 * Su producto siempre debe ser el tamaño del vector (arreglo).
 */
#define NUM_BLOCKS  4
#define THREADS_PER_BLOCK 256

/* Main routine */
int main(int argc, char *argv[])
{
    int *a, *b, *c; /* Arreglos del CPU */
    int *d_a, *d_b, *d_c;/* Arreglos del GPU */

    int i;
    size_t sz = ARRAY_SIZE * sizeof(int);

    /*
     * Reservar memoria en el cpu
     */
    a = (int *) malloc(sz);
    b = (int *) malloc(sz);
    c = (int *) malloc(sz);

    /*
     * Parte 1A:Reservar memoria en el GPU
     */
    hipMalloc((void**) &d_a, sz);
    hipMalloc((void**) &d_b, sz);
    hipMalloc((void**) &d_c, sz);

    /* inicialización */
    for (i = 0; i < ARRAY_SIZE; i++) {
        a[i] = i;
        b[i] = ARRAY_SIZE - i;
        c[i] = 0;
    }

    /* Parte 1B: Copiar los vectores del CPU al GPU */
    hipMemcpy(d_a, a, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sz, hipMemcpyHostToDevice);

    /* run the kernel on the GPU */
    /* Parte 2A: Configurar y llamar los kernels */
    dim3 dimGrid(NUM_BLOCKS, 1, 1);
    dim3 dimBlock(THREADS_PER_BLOCK / NUM_BLOCKS, 1, 1);
    //vect_add<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    vect_add_multiblock<<dimGrid, dimBlock>>(d_a, d_b, d_c);

    /* Esperar a que todos los threads acaben y checar por errores */
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* Part 1C: copiar el resultado de nuevo al CPU */
    hipMemcpy(c, d_c, sz, hipMemcpyDeviceToHost);

    checkCUDAError("memcpy");

    /* print out the result */
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", c[i]);
    }
    printf("\n\n");

    /* Parte 1D: Liberar los arreglos */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
